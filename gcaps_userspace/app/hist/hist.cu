#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <assert.h>
#include <fcntl.h>
#include <sys/ioctl.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <sched.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include <linux/nvgpu.h>
#include "hist.h"
#include <common/include/support.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BLOCK_SIZE 512

__global__ void histogram_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {
	__dummy_kernel_prologue(50);

	for (unsigned int i = 0; i < 10; i++) {
		unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
		int stride = blockDim.x * gridDim.x;
		for (unsigned int j = tid; j < num_elements; j += stride) {
			// by default, the randomly generated value should be in range (0, 4095)
			int position = input[j];
			if (position >= 0 && position <= num_bins - 1) {
				atomicAdd(&(bins[position]), 1);
			}
		}
	}
	__dummy_kernel_epilogue();
}

Hist::Hist(unsigned int n_elements, unsigned int n_bins, int fd_,  bool sync_mode_, bool ioctl_enabled_, bool suspension_) {
	num_elements = n_elements;
	num_bins = n_bins;
	fd = fd_;
	sync_mode = sync_mode_;

	ioctl_enabled = ioctl_enabled_;
	if (suspension_ == true) {
		event_flags |= hipEventBlockingSync;
    }else {
        event_flags = hipEventDisableTiming;
        // event_flags = hipEventDefault;
    }
}

void Hist::taskInit() {
	hipInit(0);
    hipDeviceGet(&device, 0);
    hipCtxCreate(&ctx, 0, device);

	if (event_flags != 0) {
        checkCudaErrors(hipEventCreateWithFlags(&start, event_flags));
        checkCudaErrors(hipEventCreateWithFlags(&stop, event_flags));
    }else {
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
    }
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	checkCudaErrors(hipHostMalloc((void **)&in_h, num_elements * sizeof(unsigned int)));
	checkCudaErrors(hipHostMalloc((void **)&bins_h, num_bins * sizeof(unsigned int)));
	for (unsigned int i = 0; i < num_elements; i++) {
		in_h[i] = rand() % num_bins;
	}

	checkCudaErrors(hipMalloc((void**)&in_d, num_elements * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc((void**)&bins_d, num_bins * sizeof(unsigned int)));
}

void Hist::taskCallback(int insId, int nIter) {
	const unsigned int GRID_SIZE = (int)ceil((float(num_elements)) / BLOCK_SIZE);
	dim3 DimGrid = dim3(GRID_SIZE);
	dim3 DimBlock = dim3(BLOCK_SIZE);

	gcapsGpuSegBegin(fd, getpid(), sync_mode, ioctl_enabled);
	checkCudaErrors(hipMemcpyAsync(in_d, in_h, num_elements * sizeof(unsigned int), hipMemcpyHostToDevice, stream));
	checkCudaErrors(hipMemsetAsync(bins_d, 0, num_bins * sizeof(unsigned int), stream));
	
	histogram_kernel <<<DimGrid, DimBlock, 0, stream>>> (in_d, bins_d, num_elements, num_bins);

	__event_record_start(insId, nIter);
	histogram_kernel <<<DimGrid, DimBlock, 0, stream>>> (in_d, bins_d, num_elements, num_bins);
	__event_record_stop(insId, nIter);
	
	checkCudaErrors(hipMemcpyAsync(bins_h, bins_d, num_bins * sizeof(unsigned int), hipMemcpyDeviceToHost, stream));
	gcapsGpuSegEnd(fd, getpid(), sync_mode, stream, ioctl_enabled);
}

void Hist::taskFinish() {
	checkCudaErrors(hipHostFree(in_h));
	checkCudaErrors(hipHostFree(bins_h));
	checkCudaErrors(hipFree(in_d));
	checkCudaErrors(hipFree(bins_d));
	hipCtxDestroy(ctx);
}

void Hist::recordPriority(int priority) {
	this->prio = priority;
}